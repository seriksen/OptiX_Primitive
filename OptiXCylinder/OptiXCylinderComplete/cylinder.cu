#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_aabb_namespace.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>

using namespace optix;

// Communication Variables
rtDeclareVariable(float3, cylinder_p, , );
rtDeclareVariable(float3, cylinder_q, , );
rtDeclareVariable(float4, cylinder_r, , );
rtDeclareVariable(float3, cylinder_min, , );
rtDeclareVariable(float3, cylinder_max, , );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float3, texcoord, attribute texcoord, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );

// static __device__ float3 cylindernormal(float t) //float t, float3 t0, float3
// t1)
//{
//  return t;
// float3 neg = make_float3(t==t0.x?1:0, t==t0.y?1:0, t==t0.z?1:0);
// float3 pos = make_float3(t==t1.x?1:0, t==t1.y?1:0, t==t1.z?1:0);
// return pos-neg;
//}

/*
 * Cylinder Intersection
 * NOTE: Based on RTCD - Christer Ericson so cylinder origin is at P.
 * Has been moved to a ray rather than segment
 *
 * Essentially it's comparing the cylinder frame to the ray frame
 *
 * Define cylinder as P,Q,r                Define ray as A,B
 * <--r-|
 * +----Q----*                          A -------------- B
 * |         |
 * |         X
 * |         |
 * +----P----+
 *
 * If X is a point on the cylinder surface then
 * (v - w) . (v - w) - r2 = 0 (eq 1)
 * where;
 * v = X - P, d = Q - P, w = ((v.d)/(d.d)) . d
 * In words;
 * - v = Distance between intersection point and point P
 * - d = height of cylinder (the axis for the calculations)
 * - w = distance of v along d axis
 *
 *
 * Intersection
 *       +---------+             L(t) = A + t(B-A)
 *       |         |
 *   A---|---------|---B
 *       |         |
 *       +---------+
 * Intersection defined as L(t) = X so solve for t
 *
 * After some rearranging we eq 1 becomes
 * (n.n - (n.d)^2 / (d.d))t^2 + 2(m.n - (n.d)(m.d)/(d.d))t
 *  + m.m - (m.d)^2 / (d.d) - r^2 = 0
 * Where m = A - P and n = B - A (from v = L(t) - P)
 * In words;
 * - n = ray direction
 * - m = ray position relative to point P
 *
 * This is what needs to be solved
 *
 * Alternatively can be written as
 * ((d.d)(n.n) - (n.d)^2)t^2 + 2((d.d)(m.n) - (n.d)(m.d))t
 *  + (d.d)((m.m)- r^2) - (m.d)^2 = 0
 *
 * So a quadratic in the form of at^2 + 2bt + c = 0
 * where
 *      a = (d.d)(n.n) - (n.d)^2         = (d x n).(d x n)
 *      b = (d.d)(m.n) - (n.d)(m.d)      = (d x m).(d x n)
 *      c = (d.d)((m.m) - r^2) - (m.d)^2 = (d x m).(d x m) - (d.d)r^2
 *
 * Key points
 * If a = 0 -> d and n are parallel
 * a > 0
 * If c < 0 -> intersect is inside cylinder surface
 * If c > 0 -> intersect is outside cylinder surface
 *
 * Solve using standard formula: t = (-b +/- sqrt(b^2 - ac)) / (a)
 *
 * Case: b^2 - ac < 0
 *  - No roots
 *  - No intersection
 *
 * Case: B^2 - ac > 0
 *  - Two roots
 *  - root1 (smaller) = value where line enters cylinder
 *  - root2 (larger) = value where line exists cylinder
 *  - Can get away with only calculating one here
 *
 * Ray could intersect with endcaps (P and Q)
 *
 * Case: P-endcap
 *  - ray is outside plane P if;
 *      (L(t) - P).d < 0 -> (m.d) + t(n.d) < 0
 *      So don't need to test against P in this case
 *  - ray is outside cylinder if;
 *    n.d <= 0
 *    -> L(t) points away from P
 *    So don't need to test against P in this case
 *  - Only need to test against P if;
 *    n.d > 0
 *    Need to test against P
 *  - Test against P
 *     (X - P).d = 0 -> t = - (m.d) / n.d
 *     (L(t) - P).(L(t) - P) <= r^2
 *
 * Case: Q-endcap
 *  - ray is outside plane Q if;
 *    (L(t) - P).d > d.d -> (m.d) + t(n.d) > d.d
 *    So don't need to test against Q
 *  - Only need to test against Q of;
 *    n.d < 0
 *  - Test against Q
 *    (X - Q).d = 0 -> t = ((d.d) - (m.d))/(n.d)
 *    (L(t) - Q).(L(t) - Q) <= r^2
 *
 * TODO: Add graphical representation of d,n,m,etc...
 *
 * intersection implementation
 * - 2 checks
 *   - endcaps
 *   - infinite cylinder
 * - endcaps
 *   - check for intersection with P and Q if ray origin is outside cylinder
 *
 * What's included
 * - End caps handling (P and Q)
 * - infinite cylinder handling
 *    - t root 1
 *    - t root 2
 * - Cases where Ray origin is inside cylinder
 */
RT_PROGRAM void intersect(int) {
  // Cylinder properties from RT variables

  float3 p = cylinder_p;
  float3 q = cylinder_q;
  float r = cylinder_r.w;

  float3 d = make_float3(0.f, 0.f, q.z - p.z); // cylinder z

  // Ray information
  float3 m = ray.origin - p; // ray origin relative to P
  float3 n = ray.direction;
  float t;

  // Initial dot products
  float mm = dot(m, m);
  float nn = dot(n, n);
  float dd = dot(d, d);
  float md = dot(m, d);
  float nd = dot(n, d);
  float mn = dot(m, n);

  // at^2 + bt + c = 0
  float a = dd * nn - nd * nd;
  float b = dd * mn - nd * md;
  float c = dd * (mm - r * r) - md * md;

  //***************
  // Test Endcaps
  //***************

  // Below P
  if (md < 0.0f && md + nd < 0.0f)
    return;
  // Above Q
  if (md > dd && md + nd > dd)
    return;

  // If a is parallel to cylinder
  if (fabs(a) < 1e-6f) {

    // outside of cylinder
    if (c > 0.f)
      return;

    // If still in, means ray intersects one of the endcaps

    // Intersects P endcap
    if (md < 0.f) {
      t = -mn / nn;
      if (rtPotentialIntersection(t)) {
        shading_normal = geometric_normal = -normalize(d);
        rtReportIntersection(0);
      }
    }
    // Intersect Q endcap
    else if (md > dd) {
      t = (nd - mn) / nn;
      if (rtPotentialIntersection(t)) {
        shading_normal = geometric_normal = -normalize(d);
        rtReportIntersection(0);
      }
    }
    // Ray origin is inside cylinder
    else {
      t = 0.f;
      if (rtPotentialIntersection(t)) {
        shading_normal = geometric_normal = -normalize(d);
        rtReportIntersection(0);
      }
    }
    return;
  }

  //************************
  // Test Infinite Cylinder
  //************************

  // Define some more things
  float disc = b * b - a * c;
  float t_root1;
  float t_root2;
  float radius_check;

  // Has no roots and no intersection
  if (disc < 0.f)
    return;

  // root 1
  t_root1 = (-b - sqrtf(disc)) / a;
  float3 root1_pos = ray.origin + t_root1 * ray.direction;

  // Intersection inside cylinder
  if (md + t_root1 * nd > 0.f && md + t_root1 * nd < dd) {
    if (rtPotentialIntersection(t_root1)) {
      shading_normal = geometric_normal = normalize(d);
      rtReportIntersection(0);
    }
  }
  // Intersection outside cylinder P side
  else if (md + t_root1 * nd < 0.f) {
    // Ray is going away from endcap
    if (nd <= 0.f)
      return;

    t = -md / nd; // P endcap
    radius_check = (mm - r * r) + t * (2.f * mn + t * nn);
    if (radius_check < 0.f) {
      if (rtPotentialIntersection(t)) {
        shading_normal = geometric_normal = normalize(d);
        rtReportIntersection(0);
      }
    }
  }
  // Intersection outside cylinder Q side
  else if (md + t_root1 * nd > dd) {
    // Ray is going away from endcap
    if (nd >= 0.f)
      return;

    t = (dd - md) / nd; // Q endcap
    radius_check =
        (mm - r * r) + dd - 2.0f * md + t * (2.f * (mn - nd) + t * nn);
    if (radius_check < 0.f) {
      if (rtPotentialIntersection(t)) {
        shading_normal = geometric_normal = -normalize(d);
        rtReportIntersection(0);
      }
    }
  }
  return;

  // root 1
  t_root2 = (-b + sqrtf(disc)) / a;
  float3 root1_pos = ray.origin + t_root2 * ray.direction;

  // Intersection inside cylinder
  if (md + t_root2 * nd > 0.f && md + t_root2 * nd < dd) {
    if (rtPotentialIntersection(t_root2)) {
      shading_normal = geometric_normal = normalize(d);
      rtReportIntersection(0);
    }
  }
    // Intersection outside cylinder P side
  else if (md + t_root2 * nd < 0.f) {
    // Ray is going away from endcap
    if (nd <= 0.f)
      return;

    t = -md / nd; // P endcap
    radius_check = (mm - r * r) + t * (2.f * mn + t * nn);
    if (radius_check < 0.f) {
      if (rtPotentialIntersection(t)) {
        shading_normal = geometric_normal = normalize(d);
        rtReportIntersection(0);
      }
    }
  }
    // Intersection outside cylinder Q side
  else if (md + t_root2 * nd > dd) {
    // Ray is going away from endcap
    if (nd >= 0.f)
      return;

    t = (dd - md) / nd; // Q endcap
    radius_check =
        (mm - r * r) + dd - 2.0f * md + t * (2.f * (mn - nd) + t * nn);
    if (radius_check < 0.f) {
      if (rtPotentialIntersection(t)) {
        shading_normal = geometric_normal = -normalize(d);
        rtReportIntersection(0);
      }
    }
  }
}

RT_PROGRAM void bounds(int, float result[6]) {
  optix::Aabb *aabb = (optix::Aabb *)result;
  aabb->set(cylinder_min, cylinder_max);
}
