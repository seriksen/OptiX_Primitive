#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include <optixu/optixu_aabb_namespace.h>

using namespace optix;

// Communication Variables
rtDeclareVariable(float3, cylinder_p, , );
rtDeclareVariable(float3, cylinder_q, , );
rtDeclareVariable(float4, cylinder_r, , );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float3, texcoord, attribute texcoord, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );

//static __device__ float3 cylindernormal(float t) //float t, float3 t0, float3 t1)
//{
//  return t;
  //float3 neg = make_float3(t==t0.x?1:0, t==t0.y?1:0, t==t0.z?1:0);
  //float3 pos = make_float3(t==t1.x?1:0, t==t1.y?1:0, t==t1.z?1:0);
  //return pos-neg;
//}

/*
 * Cylinder Intersection
 * NOTE: Follows intersection maths described in RTCD - Christer Ericson so
 * cylinder origin is at P.
 *
 * Essentially it's comparing the cylinder frame to the ray frame
 *
 * Define cylinder as P,Q,r                Define ray as A,B
 * <--r-|
 * +----Q----*                          A -------------- B
 * |         |
 * |         X
 * |         |
 * +----P----+
 *
 * If X is a point on the cylinder surface then
 * (v - w) . (v - w) - r2 = 0 (eq 1)
 * where;
 * v = X - P, d = Q - P, w = ((v.d)/(d.d)) . d
 *
 * Intersection
 *       +---------+             L(t) = A + t(B-A)
 *       |         |
 *   A---|---------|---B
 *       |         |
 *       +---------+
 * Intersection defined as L(t) = X so solve for t
 *
 * After some rearranging we eq 1 becomes
 * (n.n - (n.d)^2 / (d.d))t^2 + 2(m.n - (n.d)(m.d)/(d.d))t
 *  + m.m - (m.d)^2 / (d.d) - r^2 = 0
 * Where m = A - P and n = B - A (from v = L(t) - P)
 *
 * This is what needs to be solved
 *
 * Alternatively can be written as
 * ((d.d)(n.n) - (n.d)^2)t^2 + 2((d.d)(m.n) - (n.d)(m.d))t
 *  + (d.d)((m.m)- r^2) - (m.d)^2 = 0
 *
 * So a quadratic in the form of at^2 + 2bt + c = 0
 * where
 *      a = (d.d)(n.n) - (n.d)^2         = (d x n).(d x n)
 *      b = (d.d)(m.n) - (n.d)(m.d)      = (d x m).(d x n)
 *      c = (d.d)((m.m) - r^2) - (m.d)^2 = (d x m).(d x m) - (d.d)r^2
 *
 * Key points
 * If a = 0 -> d and n are parallel
 * a > 0
 * If c < 0 -> intersect is inside cylinder surface
 * If c > 0 -> intersect is outside cylinder surface
 *
 * Solve using standard formula: t = (-b +/- sqrt(b^2 - ac)) / (a)
 *
 * Case: b^2 - ac < 0
 *  - No roots
 *  - No intersection
 *
 * Case: B^2 - ac > 0
 *  - Two roots
 *  - root1 (smaller) = value where line enters cylinder
 *  - root2 (larger) = value where line exists cylinder
 *
 * Ray could intersect with endcaps (P and Q)
 *
 * Case: P-endcap
 *  - ray is outside plane P if;
 *      (L(t) - P).d < 0 -> (m.d) + t(n.d) < 0
 *      So don't need to test against P in this case
 *  - ray is outside cylinder if;
 *    n.d <= 0
 *    -> L(t) points away from P
 *    So don't need to test against P in this case
 *  - Only need to test against P if;
 *    n.d > 0
 *    Need to test against P
 *  - Test against P
 *     (X - P).d = 0 -> t = - (m.d) / n.d
 *     (L(t) - P).(L(t) - P) <= r^2
 *
 * Case: Q-endcap
 *  - ray is outside plane Q if;
 *    (L(t) - P).d > d.d -> (m.d) + t(n.d) > d.d
 *    So don't need to test against Q
 *  - Only need to test against Q of;
 *    n.d < 0
 *  - Test against Q
 *    (X - Q).d = 0 -> t = ((d.d) - (m.d))/(n.d)
 *    (L(t) - Q).(L(t) - Q) <= r^2
 *
 * TODO: Add graphical representation of d,n,m,etc...
 *
 * intersection implementation
 * - 2 checks
 *   - endcaps
 *   - infinite cylinder
 * - endcaps
 *   - check for intersection with P and Q if ray origin is outside cylinder
 *
 */
RT_PROGRAM void intersect(int) {
  // Cylinder properties from RT variables
  float3 position = cylinder_p;  // 0,0,-169.  <-- P: point on axis at base of cylinder

  float zmin = cylinder_p.z ;  // using bbox z-range
  float zmax = cylinder_q.z ;

  float clipped_sizeZ = zmax - zmin ;
  float radius = cylinder_r.w ;


  //rtPrintf("intersect_ztubs position %10.4f %10.4f %10.4f \n", position.x, position.y, position.z );
  //rtPrintf("intersect_ztubs flags %d PCAP %d QCAP %d \n", flags, PCAP, QCAP);

  float3 m = ray.origin - position ;                  // ray origin in cylinder P-frame
  float3 n = ray.direction ;
  float3 d = make_float3(0.f, 0.f, clipped_sizeZ );   // cylinder axis

  float rr = radius*radius ;
  float3 dnorm = normalize(d);


  float mm = dot(m, m) ;
  float nn = dot(n, n) ;
  float dd = dot(d, d) ;
  float nd = dot(n, d) ;
  float md = dot(m, d) ;
  float mn = dot(m, n) ;
  float k = mm - rr ;

  // quadratic coefficients of t,     a tt + 2b t + c = 0
  float a = dd*nn - nd*nd ;
  float b = dd*mn - nd*md ;
  float c = dd*k - md*md ;

  float disc = b*b-a*c;

  // axial ray endcap handling
  if(fabs(a) < 1e-6f)
  {
    if(c > 0.f) return ;    // ray starts and ends outside cylinder
    if(md < 0.f)    // ray origin on P side
    {
      float t = -mn/nn ;  // P endcap
      if( rtPotentialIntersection(t) )
      {
        shading_normal = geometric_normal = -dnorm  ;
        rtReportIntersection(0);
      }
    }
    else if(md > dd) // ray origin on Q side
    {
      float t = (nd - mn)/nn ;  // Q endcap
      if( rtPotentialIntersection(t) )
      {
        shading_normal = geometric_normal = dnorm ;
        rtReportIntersection(0);
      }
    }
    else    // md 0->dd, ray origin inside
    {
      if( nd > 0.f) // ray along +d
      {
        float t = -mn/nn ;    // P endcap from inside
        if( rtPotentialIntersection(t) )
        {
          shading_normal = geometric_normal = dnorm  ;
          rtReportIntersection(0);
        }
      }
      else  // ray along -d
      {
        float t = (nd - mn)/nn ;  // Q endcap from inside
        if( rtPotentialIntersection(t) )
        {
          shading_normal = geometric_normal = -dnorm ;
          rtReportIntersection(0);
        }
      }
    }
    return ;   // hmm
  }

  if(disc > 0.0f)  // intersection with the infinite cylinder
  {
    float sdisc = sqrtf(disc);

    float root1 = (-b - sdisc)/a;

    // m:ray.origin-position
    // n:ray.direction

    float ad1 = md + root1*nd ;        // axial coord of intersection point (* sizeZ)
    float3 P1 = ray.origin + root1*ray.direction ;

    if( ad1 > 0.f && ad1 < dd )  // intersection inside cylinder range
    {
      if( rtPotentialIntersection(root1) )
      {
        float3 N  = (P1 - position)/radius  ;
        N.z = 0.f ;

        //rtPrintf("intersect_ztubs r %10.4f disc %10.4f sdisc %10.4f root1 %10.4f P %10.4f %10.4f %10.4f N %10.4f %10.4f \n",
        //    radius, disc, sdisc, root1, P1.x, P1.y, P1.z, N.x, N.y );

        shading_normal = geometric_normal = normalize(N) ;
        rtReportIntersection(0);
      }
    }
    else if( ad1 < 0.f ) //  intersection outside cylinder on P side
    {
      if( nd <= 0.f ) return ; // ray direction away from endcap
      float t = -md/nd ;   // P endcap
      float checkr = k + t*(2.f*mn + t*nn) ; // bracket typo in book 2*t*t makes no sense
      if ( checkr < 0.f )
      {
        if( rtPotentialIntersection(t) )
        {
          shading_normal = geometric_normal = -dnorm  ;
          rtReportIntersection(0);
        }
      }
    }
    else if( ad1 > dd  ) //  intersection outside cylinder on Q side
    {
      if( nd >= 0.f ) return ; // ray direction away from endcap
      float t = (dd-md)/nd ;   // Q endcap
      float checkr = k + dd - 2.0f*md + t*(2.f*(mn-nd)+t*nn) ;
      if ( checkr < 0.f )
      {
        if( rtPotentialIntersection(t) )
        {
          shading_normal = geometric_normal = dnorm  ;
          rtReportIntersection(0);
        }
      }
    }


    float root2 = (-b + sdisc)/a;     // far root : means are inside (always?)
    float ad2 = md + root2*nd ;        // axial coord of far intersection point
    float3 P2 = ray.origin + root2*ray.direction ;


    if( ad2 > 0.f && ad2 < dd )  // intersection from inside against wall
    {
      if( rtPotentialIntersection(root2) )
      {
        float3 N  = (P2 - position)/radius  ;
        N.z = 0.f ;

        shading_normal = geometric_normal = -normalize(N) ;
        rtReportIntersection(0);
      }
    }
    else if( ad2 < 0.f ) //  intersection from inside to P endcap
    {
      float t = -md/nd ;   // P endcap
      float checkr = k + t*(2.f*mn + t*nn) ; // bracket typo in book 2*t*t makes no sense
      if ( checkr < 0.f )
      {
        if( rtPotentialIntersection(t) )
        {
          shading_normal = geometric_normal = dnorm  ;
          rtReportIntersection(0);
        }
      }
    }
    else if( ad2 > dd ) //  intersection from inside to Q endcap
    {
      float t = (dd-md)/nd ;   // Q endcap
      float checkr = k + dd - 2.0f*md + t*(2.f*(mn-nd)+t*nn) ;
      if ( checkr < 0.f )
      {
        if( rtPotentialIntersection(t) )
        {
          shading_normal = geometric_normal = -dnorm  ;
          rtReportIntersection(0);
        }
      }
    }
  }
}

/*
  float3 p = cylinder_p;
  float3 q = cylinder_q;
  float r = cylinder_r.w;

  float3 d = make_float3(0.f,0.f,q.z - p.z); // cylinder z
  if (rtPotentialIntersection(0.01f)) {
    shading_normal = geometric_normal = make_float3(0.f, 1.0f, 0.f);
    rtReportIntersection(0);
  }
  return;
  // Ray information
  float3 m = ray.origin - p; // ray origin relative to P
  float3 n = ray.direction;

  // Initial dot products
  // TODO: Move them all together
  float md = dot(m, d);
  float nd = dot(n, d);
  float dd = dot(d, d);

  //***************
  // Test Endcaps
  //***************

  // Below P
  if (md < 0.0f && md + nd < 0.0f)
    return;
  // Above Q
  if (md > dd && md + nd > dd)
    return;

  // More dot products
  float nn = dot(n,n);
  float mn = dot(m,n);
  float mm = dot(m,m);
  float a = dd * nn - nd * nd;
  float c = dd * (mm - r*r) - md * md;

  // Also define t
  float t;

  // If a is parallel to cylinder
  if (fabs(a) < 1e-6f) {

    // outside of cylinder
    if (c > 0.f) return;

    // If still in, means ray intersects

    // Intersects P endcap
    if (md < 0.f) {
      t = - mn / nn;
      if (rtPotentialIntersection(t)) {
        shading_normal = geometric_normal = -normalize(d);
        rtReportIntersection(0);
      }
    }
    // Intersect Q endcap
    else if (md > dd) {
      t = (nd - mn) / nn;
      if (rtPotentialIntersection(t)) {
        shading_normal = geometric_normal = normalize(d);
        rtReportIntersection(0);
      }
    }
    // Ray origin is inside cylinder
    else {
      // going to say that the ray counts as a miss for now
      //t = 0.f;
      return;
    }
  }

  //************************
  // Test Infinite Cylinder
  //************************

  // Define some more things
  float b = dd * mn - nd * md;
  float disc = b*b - a*c;

  // Has no roots
  if (disc < 0.f) return;


  t = (-b - sqrtf(disc)) / a;
  float radius_check;
  // Intersection is outside segment
  if (t < 0.f || t > 1.0f) return;

  // Intersection on P side
  if (md + t * nd < 0.f) {
    // Ray is going away from endcap
    if (nd <= 0.f) return;

    t = -md/nd; // P endcap
    radius_check = mm -r*r + t * (2.f * mn + t*nn);
    if (radius_check <= 0.f) {
      if (rtPotentialIntersection(t)) {
        shading_normal = geometric_normal = -normalize(d);
        rtReportIntersection(0);
      }
    }
  }
  // Intersection on Q side
  else if (md + t * nd > dd) {
    // Ray is going away from endcap
    if (nd >= 0.f) return;

    t = (dd - md) / nd; // Q endcap
    radius_check = mm - r*r + t * (2.f * (mn - nd) + t * nn);
    if (radius_check <= 0.f) {
      if (rtPotentialIntersection(t)) {
        shading_normal = geometric_normal = normalize(d);
      }
    }
  }
  // ray intersects cylinder between the end caps
  else {
      if (rtPotentialIntersection(t)) {
        shading_normal = geometric_normal = normalize(d);
      }
  }

  // FIXME: add second root
  return;

}
*/
RT_PROGRAM void bounds (int, float result[6])
{
  optix::Aabb* aabb = (optix::Aabb*)result;
  aabb->set(cylinder_p, cylinder_q);
}
