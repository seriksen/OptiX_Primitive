#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include <optixu/optixu_aabb_namespace.h>

using namespace optix;

// Communication Variables
rtDeclareVariable(float3, cylinder_min, , );
rtDeclareVariable(float3, cylinder_max, , );
rtDeclareVariable(float3, cylinder_r, , );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float3, texcoord, attribute texcoord, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );

static __device__ float3 cylindernormal() //float t, float3 t0, float3 t1)
{
  return make_float3(0f,0f,0f);
  //float3 neg = make_float3(t==t0.x?1:0, t==t0.y?1:0, t==t0.z?1:0);
  //float3 pos = make_float3(t==t1.x?1:0, t==t1.y?1:0, t==t1.z?1:0);
  //return pos-neg;
}

// cylinder intersection
// Following intersection maths described ub RTCD - Christer Ericson
// Define
/**
 * @brief Calculate if Cylinder intersection
 * NOTE: Follows intersection maths described in RTCD - Christer Ericson so
 * cylinder origin is at P.
 *
 *
 * Define cylinder as P,Q,r                Define ray as A,B
 * <--r-|
 * +----Q----*                          A -------------- B
 * |         |
 * |         |
 * +----P----+
 *
 * If X is a point on the cylinder surface then
 * (v - w) . (v - w) - r2 = 0
 * where;
 * v = X - P, d = Q - P, w = ((v.d)/(d.d)) . d
 *
 * Intersection
 *       +---------+             L(t) = A + t(B-A)
 *       |         |
 *   A---|---------|---B
 *       |         |
 *       +---------+
 * Intersection defined as L(t) = X
 *
 */
RT_PROGRAM void intersect(int)
{
  // TODO: Add second half of intersection

  // Cylinder information
  float3 p_loc = cylinder_min; // P location
  float3 q_loc = cylinder_max; // Q location
  //float z = cylinder_max.z - cylinder_min.z;
  float r = cylinder_r.x;
  float3 d = (q_loc - p_loc) / ray.direction;

  // Ray information
  float3 m = ray.origin - p_loc; // ray origin relative to P
  float3 n = ray.direction;

  // Other vars
  bool hasIntersect{false};
  bool check_second = true;

  // Calculate dot products
  float md = dot(m,d);
  float nd = dot(n,d);
  float dd = dot(d,d);
  float nn = dot(n,n);
  float mn = dot(m,n);
  float mm = dot(m,m);
  float a = dd * nn - nd * nd;
  float k = mm - r*r;
  float c = dd * k - md * md;
  float t;

  // Test if fully outside endcaps of cylinder
  if (md < 0.0f && md + nd < 0.0f) {
    // Not in cylinder
    // Below P
    return;
  }
  else if (md > dd && md + nd > dd) {
    // Not in cylinder
    // Above P and Q
    return;
  }
  // Is within endcaps
  else if (fabs(a) < 1e-6f) {
    // ray is parrallel to cylinder axis
    if (c > 0.f) {
      // Not in cyclinder
      // 'a' is outside cylinder
      return;
    }
    // Check if endcap intersect
    if (md < 0.f) {
      // Intersect with P
      t = -mn / nn;
      hasIntersect = true;
    } else if (md > dd) {
      // Intersect with Q
      t = (nd - mn) / nn;
      hasIntersect = true;
    } else {
      // 'a' is inside cylinder
      t = 0.0f;
      hasIntersect = true;
    }
    check_second = false;
  }
  if (check_second) {
    float b = dd * mn - nd * md;
    float discr = b * b - a * c;
    if (discr < 0.0f) {
      // No roots and no intersection
      return;
    }
    t = (-b - sqrtf(discr)) / a;
    if (t < 0.0f || t > 1.0f) {
      // root is outside segment so no intersection
      return;
    }
    if (md + t * nd < 0.0f) {
      // Intersection is outside cylinder of P
      if (nd <= 0.0f) {
        // Segment pointing away from endcap
        return;
      }
      t = -md / nd;
      if ((k + 2 * t * (mn + t * nn)) <= 0.0f) {
        hasIntersect = true;
      }
    } else if (md + t * nd > dd) {
      // Intersection is outside cylinder on Q side
      if (nd >= 0.0f) {
        t = (dd - md) / nd;
        if ((k + dd - 2 * md + t * (2 * (mn - nd) + t * nn)) <= 0.0f) {
          hasIntersect = true;
        }
      }
    }
  }
  if (hasIntersect) {
    if (rtPotentialIntersection(t)) {
      texcoord = make_float3( 0.0f );
      shading_normal = geometric_normal = cylindernormal(); //cylindernormal(t, p_loc, q_loc);
      printf(t);
      rtReportIntersection(0);
      }
    }
  }

RT_PROGRAM void bounds (int, float result[6])
{
  optix::Aabb* aabb = (optix::Aabb*)result;
  aabb->set(cylinder_min, cylinder_max);
}
