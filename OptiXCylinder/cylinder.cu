#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include <optixu/optixu_aabb_namespace.h>

using namespace optix;

// Communication Variables
rtDeclareVariable(float3, cylinder_min, , );
rtDeclareVariable(float3, cylinder_max, , );
rtDeclareVariable(float3, cylinder_r, , );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float3, texcoord, attribute texcoord, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );

static __device__ float3 cylindernormal(float t, float3 t0, float3 t1)
{
  float3 neg = make_float3(t==t0.x?1:0, t==t0.y?1:0, t==t0.z?1:0);
  float3 pos = make_float3(t==t1.x?1:0, t==t1.y?1:0, t==t1.z?1:0);
  return pos-neg;
}

// cylinder intersection
// Following intersection maths described ub RTCD - Christer Ericson
// Define
/**
 * @brief Calculate if Cylinder intersection
 * NOTE: Follows intersection maths described in RTCD - Christer Ericson so
 * cylinder origin is at P.
 *
 *
 * Define cylinder as P,Q,r                Define ray as A,B
 * <--r-|
 * +----Q----*                          A -------------- B
 * |         |
 * |         |
 * +----P----+
 *
 * If X is a point on the cylinder surface then
 * (v - w) . (v - w) - r2 = 0
 * where;
 * v = X - P, d = Q - P, w = ((v.d)/(d.d)) . d
 *
 * Intersection
 *       +---------+             L(t) = A + t(B-A)
 *       |         |
 *   A---|---------|---B
 *       |         |
 *       +---------+
 * Intersection defined as L(t) = X
 *
 */
RT_PROGRAM void intersect(int)
{
  // Cylinder information
  float3 p_loc = cylinder_min; // P location
  float3 q_loc = cylinder_max; // Q location
  float z = q_loc.z - p_loc.z;
  float r = cylinder_r.x;
  float3 d = q_loc - p_loc;

  // Ray information
  float3 m = ray.origin - p_loc; // ray origin relative to P
  float3 n = ray.direction;

  // Other vars
  bool check_second = true;

  // Calculate dot products
  float md = dot(m,d);
  float nd = dot(n,d);
  float dd = dot(d,d);
  float nn = dot(n,n);
  float mn = dot(m,n);
  float mm = dot(m,m);
  float a = dd * nn - nd * nd;
  float k = mm - r*r;
  float c = dd * k - md * md;
  float t;

  // Test if fully outside endcaps of cylinder
  if (md < 0.0f && md + nd < 0.0f) {
    // Not in cylinder
    // Below P
    return;
  }
  else if (md > dd && md + nd > dd) {
    // Not in cylinder
    // Above P and Q
    return;
  }
  // Is within endcaps
  else if (fabs(a) < 1e-6f) {
    if (c > 0.f) {
      // Not in cyclinder
      // 'a' is outside cylinder
      return;
    }
    // Check if endcap intersect
    if (md < 0.f) {
      // Intersect with P
      t = -mn / nn;
    }
    else if (md > dd) {
      // Intersect with Q
      t = (nd - mn) / nn;
    }
    else {
      // 'a' is inside cylinder
      if (rtPotentialIntersection()) {
        shading_normal = geometric_normal = cylindernormal(m, p_loc, q_loc);
        if(rtReportIntersection(0)) {
          check_second = false;
        }
      }
    }

  }
}

RT_PROGRAM void bounds (int, float result[6])
{
  optix::Aabb* aabb = (optix::Aabb*)result;
  aabb->set(cylinder_min, cylinder_max);
}
