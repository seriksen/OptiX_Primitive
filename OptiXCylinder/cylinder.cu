#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include <optixu/optixu_aabb_namespace.h>

using namespace optix;

// Communication Variables
rtDeclareVariable(float3, cylinder_min, , );
rtDeclareVariable(float3, cylinder_max, , );
rtDeclareVariable(float3, cylinder_r, , );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float3, texcoord, attribute texcoord, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );

static __device__ float3 cylindernormal(float t, float3 t0, float3 t1)
{
  float3 neg = make_float3(t==t0.x?1:0, t==t0.y?1:0, t==t0.z?1:0);
  float3 pos = make_float3(t==t1.x?1:0, t==t1.y?1:0, t==t1.z?1:0);
  return pos-neg;
}

// cylinder intersection
// Following intersection maths described ub RTCD - Christer Ericson
// Define
/**
 * @brief Calculate if Cylinder intersection
 * NOTE: Follows intersection maths described in RTCD - Christer Ericson so
 * cylinder origin is at P.
 *
 *
 * Define cylinder as P,Q,r                Define ray as A,B
 * <--r-|
 * +----Q----*                          A -------------- B
 * |         |
 * |         |
 * +----P----+
 *
 * If X is a point on the cylinder surface then
 * (v - w) . (v - w) - r2 = 0
 * where;
 * v = X - P, d = Q - P, w = ((v.d)/(d.d)) . d
 *
 * Intersection
 *       +---------+             L(t) = A + t(B-A)
 *       |         |
 *   A---|---------|---B
 *       |         |
 *       +---------+
 * Intersection defined as L(t) = X
 *
 */
RT_PROGRAM void intersect(int)
{
  float3 d = float3(0.f, 0.f, 0.25f) / ray.direction; // unit length
  float3 m = ray.origin - cylinder_min; // Relative to P
  float3 p = (cylinder_min - ray.origin)/ray.direction;
  float3 q = (cylinder_max - ray.origin)/ray.direction;
  float3 n = ray.direction;
  float r = cylinder_r.x;
  bool check_second = true;

  // Calculate variables
  float3 md = dot(m,d);
  float3 nd = dot(n,d);
  float3 dd = dot(d,d);
  float3 nn = dot(n,n);
  float3 mn = dot(m,n);
  float3 mm = dot(m,m);
  float a = dd * nn - nd * nd;
  float k = mm - r*r;
  float c = dd * k - md * md;

  // Test if fully outside endcaps of cylinder
  if (md < 0.0f && md + nd < 0.0f) {
    // Not in cylinder
    // Below P
    return;
  }
  else if (md > dd && md + nd > dd) {
    // Not in cylinder
    // Above P and Q
    return;
  }
  // Is within endcaps
  else if (fabs(a) < 1e-6f) {
    if (c > 0.f) {
      // Not in cyclinder
      // 'a' is outside cylinder
      return;
    }
    // Check if endcap intersect
    if (md < 0.f) {
      // Intersect with P
      t = -mn / nn;
    }
    else if (md > dd) {
      // Intersect with Q
      t = (nd - mn) / nn;
    }
    else {
      // 'a' is inside cylinder
      if (rtPotentialIntersection()) {
        shading_normal = geometric_normal = cylindernormal(m, p, q);
        if(rtReportIntersection(0)) {
          check_second = false;
        }
      }
    }

  }
}

RT_PROGRAM void bounds (int, float result[6])
{
  optix::Aabb* aabb = (optix::Aabb*)result;
  aabb->set(cylinder_min, cylinder_max);
}
