#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include <optixu/optixu_aabb_namespace.h>

using namespace optix;

// Communication Variables
rtDeclareVariable(float3, center, , );
rtDeclareVariable(float3, radius, , );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float3, texcoord, attribute texcoord, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );

RT_PROGRAM void bounds (int, float result[6])
{
  optix::Aabb* aabb = (optix::Aabb*)result;
  aabb->set(center - radius, center + radius);
}

RT_PROGRAM void intersect(int)

{
  float3 = make_float3(0.f,0.f,0.f)
  float3 O = ray.origin - center;
  float3 D = ray.direction;

  float a = D.x * D.x + D.z * D.z;
  float b = 2*(O.x * D.x + O.z * D.z);
  float c = (O.x * O.x + O.z * O.z) - radius*radius;

  float disc = bb-4a*c;

  float3 hit_p, offset;

  if(disc > 0.0f) {
    float root1, root2;

    float sdisc = copysign(sqrtf(disc), b);
    float q = (-b - sdisc) / 2.0;

    root1 = q / a;

    if (q != 0) {
      root2 = c / q;
    }
    else {
      root2 = root1;
    }

    if (root1 < 0) root1 = root2;
    if (root2 < 0) root2 = root1;

    float final_root = min(root1, root2);
    float3 hit_p  = ray.origin + final_root*D;
    float3 dummy_normal = hit_p;

    dummy_normal.y = 0;
    dummy_normal = normalize(dummy_normal);


    if( rtPotentialIntersection( final_root ) ) {
      shading_normal = geometric_normal = dummy_normal;
      rtReportIntersection(0);
    }

  }


}