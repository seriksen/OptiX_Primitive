#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_aabb_namespace.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>

using namespace optix;

// Communication Variables
rtDeclareVariable(float4, disc_shape, , );
rtDeclareVariable(float3, disc_max, , );
rtDeclareVariable(float3, disc_min, , );
rtDeclareVariable(float4, disc_hole, , );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float3, texcoord, attribute texcoord, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );


RT_PROGRAM void intersect(int) {

  /*
   * for disc
   *                 ^ n
   *         i       |
   * --------*-------C--------------------
   *       d/        |----------r-------->|
   *       /
   *      * O
   *
   * Ray direction = d
   * Ray origin = O
   * disc centre = C
   * disc radius = r
   *
   * ray position = ray origin + time * ray direction
   * -> r(t) = O + t * d
   *
   * Ray intercepts with Disc if
   * 1. Is in plane
   * 2. Is within disc radius
   *
   * Ray intersects plane which disc is in if
   * (r(t) - C) . n = 0
   *
   * Ray is within radius if
   * | i - C | < r
   *
   * Let r(t) = i
   *
   * -> r(t) - C < r
   * -> (r(t) - C)^2 < r^2 (so handles both directions)
   *
   * In that case, t must satisfy O + td - C = 0
   *
   * t = (disc centre - ray origin) / ray direction  in normalised plane
   * -> t = (C - o) / d   . n (for normalised plane)
   *
   * So ray intercepts disc if...
   * r_sq = (r(t) - c).(r(t) - c) < disc_rr
   * = (o + td - c).(o + td - c)
   * only care about n direction and let o - c = m
   * = (m + tn) . (m + tn)
   * = mm + 2tnm + ttnn
   * = t (2nm + tnn) + mm
   * = t (2n (o-c) + tnn) + (o-c)(o-c)
   * r_sq < disc_rr
   *
   * For the hole
   * hole is in same plane so only need to worry about radius
   * this time
   * | i - C | > r to be outside of hole
   */

  // Disc properties
  float disc_r = disc_shape.w;
  float3 disc_c = make_float3(disc_shape.x,disc_shape.y,disc_shape.z);
  float3 disc_n = make_float3(0.f,0.f,1.f); // normal

  // Hole properties
  float3 hole_c = make_float3(disc_hole.x, disc_hole.y, disc_hole.z);
  float hole_r = disc_hole.w;

  // ray properties
  float3 ray_d = ray.direction;
  float3 ray_o = ray.origin;

  // t
  float disc_t = dot((disc_c - ray_o), disc_n) / dot (ray_d,disc_n);
  float t_min = 0.f;

  // check if intersects
  float disc_r_sq = disc_t * (2.f * dot((ray_o-disc_c), ray_d) + disc_t * dot(ray_d,ray_d))
               + dot(ray_o-disc_c,ray_o-disc_c);
  float disc_rr = disc_r*disc_r;

  if (disc_r_sq < disc_rr && disc_t > t_min) {
    // Now check hole
    float hole_t = dot((hole_c - ray_o), disc_n) / dot(ray_d,disc_n);
    float hole_r_sq = hole_t * ( dot((ray_o - hole_c), ray_d) + hole_t * dot(ray_d,ray_d))
                              + dot(ray_o - hole_c,ray_o - hole_c);
    float hole_rr = hole_r*hole_r;
    if (hole_r_sq > hole_rr && hole_t > t_min) {
      if (rtPotentialIntersection(hole_t)) {
        shading_normal = geometric_normal = normalize(disc_n);
        rtReportIntersection(0);
      }
    }
  }
  return;
}

RT_PROGRAM void bounds(int, float result[6]) {
  optix::Aabb *aabb = (optix::Aabb *)result;
  aabb->set(disc_min, disc_max);
}
