#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_aabb_namespace.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>

using namespace optix;

// Communication Variables
rtDeclareVariable(float4, disc_shape, , );
rtDeclareVariable(float3, disc_max, , );
rtDeclareVariable(float3, disc_min, , );
rtDeclareVariable(float4, disc_hole, , );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float3, texcoord, attribute texcoord, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );


RT_PROGRAM void intersect(int) {

  /*
   * for disc
   *                 ^ n
   *         i       |
   * --------*-------C--------------------
   *       d/        |----------r-------->|
   *       /
   *      * O
   *
   * Ray direction = d
   * Ray origin = O
   * disc centre = C
   * disc radius = r
   *
   * ray position = ray origin + time * ray direction
   * -> r(t) = O + t * d
   *
   * Ray intersects plane which disc is in if
   * (r(t) - C) . n = 0
   *
   * Ray hits disc is in plane and within radius
   * r(t) - C < r
   * -> (r(t) - C)^2 < r^2 (so handles both directions)
   *
   * t = (ray origin - disc centre) / ray direction  in normalised plane
   * -> t = (O - C).n / d.n
   */

  // Disc properties
  float r = disc_shape.w;
  float3 c = make_float3(disc_shape.x,disc_shape.y,disc_shape.z);
  float3 n = make_float3(0.f,0.f,1.f); // normal

  // Hole properties
  float3 hole_c = make_float3(disc_hole.x, disc_hole.y, disc_hole.z);
  float hole_r = disc_hole.w;

  // ray properties
  float3 d = ray.direction;
  float3 o = ray.origin;

  // t
  float t = - dot((o - c), n) / dot(d, n);
  float t_min = 0.f;

  // check if intersects
  float rt_sqrt = t * (2.f * dot((o-c), d) + t * dot(d,d)) + dot(o-c,o-c);
  float rr = r*r;

  if (rt_sqrt < rr && t > t_min) {
    // Now check hole
    float t_hole = - dot((o - hole_c), n) / dot(d, n);
    float rt_sqrt_h = t_hole * (2.f * dot((o - hole_c), d) + t * dot(d,d) + dot(o-hole_c,o-hole_c));
    float hole_rr = hole_r*hole_r;
    if (rt_sqrt > hole_rr && t_hole > t_min) {
      if (rtPotentialIntersection(t)) {
        shading_normal = geometric_normal = normalize(n);
        rtReportIntersection(0);
      }
    }
  }
  return;
}

RT_PROGRAM void bounds(int, float result[6]) {
  optix::Aabb *aabb = (optix::Aabb *)result;
  aabb->set(disc_min, disc_max);
}
