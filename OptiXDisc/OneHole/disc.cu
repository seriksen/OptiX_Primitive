#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_aabb_namespace.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>

using namespace optix;

// Communication Variables
rtDeclareVariable(float4, disc_shape, , );
rtDeclareVariable(float3, disc_max, , );
rtDeclareVariable(float3, disc_min, , );
rtDeclareVariable(float4, disc_hole, , );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float3, texcoord, attribute texcoord, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );


RT_PROGRAM void intersect(int) {

  /*
   * for disc
   *                 ^ n
   *         i       |
   * --------*-------C--------------------
   *       d/        |----------r-------->|
   *       /
   *      * O
   *
   * Ray direction = d
   * Ray origin = O
   * disc centre = C
   * disc radius = r
   *
   * ray position = ray origin + time * ray direction
   * -> r(t) = O + t * d
   *
   * Ray intercepts with Disc if
   * 1. Is in plane
   * 2. Is within disc radius
   *
   * Ray intersects plane which disc is in if
   * (r(t) - C) . n = 0
   *
   * Ray is within radius if
   * | i - C | < r
   *
   * Let r(t) = i
   *
   * -> r(t) - C < r
   * -> (r(t) - C)^2 < r^2 (so handles both directions)
   *
   * In that case, t must satisfy O + td - C = 0
   *
   * t = (disc centre - ray origin) / ray direction  in normalised plane
   * -> t = (C - o) / d   . n (for normalised plane)
   */

  // Disc properties
  float r = disc_shape.w;
  float3 c = make_float3(disc_shape.x,disc_shape.y,disc_shape.z);
  float3 n = make_float3(0.f,0.f,1.f); // normal

  // Hole properties
  float3 hole_c = make_float3(disc_hole.x, disc_hole.y, disc_hole.z);
  float hole_r = disc_hole.w;

  // ray properties
  float3 d = ray.direction;
  float3 o = ray.origin;

  // t
  float t = dot((c - o), n) / dot (d,n);
  float t_min = 0.f;

  // check if intersects
  // r_sq = (r(t) - c).(r(t) - c) < disc_rr
  // = (o + td - c).(o + td - c)
  // only care about n direction and let o - c = m
  // = (m + tn) . (m + tn)
  // = mm + 2tnm + ttnn
  // = t (2nm + tnn) + mm
  // = t (2n (o-c) + tnn) + (o-c)(o-c)

  float r_sq = t * (2.f * dot((o-c), d) + t * dot(d,d)) + dot(o-c,o-c);
  float rr = r*r;

  if (rt_sq < rr && t > t_min) {
    // Now check hole
    float t_hole = - dot((o - hole_c), n) / dot(d, n);
    float rt_sq_h = t_hole * (2.f * dot((o - hole_c), d) + t_hole * dot(d,d))
                              + dot(o - hole_c,o - hole_c));
    float hole_rr = hole_r*hole_r;
    if (rt_sq_h > hole_rr && t_hole > t_min) {
      if (rtPotentialIntersection(t)) {
        shading_normal = geometric_normal = normalize(n);
        rtReportIntersection(0);
      }
    }
  }
  return;
}

RT_PROGRAM void bounds(int, float result[6]) {
  optix::Aabb *aabb = (optix::Aabb *)result;
  aabb->set(disc_min, disc_max);
}
