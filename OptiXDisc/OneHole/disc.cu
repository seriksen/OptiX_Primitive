#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_aabb_namespace.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>

using namespace optix;

// Communication Variables
rtDeclareVariable(float4, disc_shape, , );
rtDeclareVariable(float3, disc_max, , );
rtDeclareVariable(float3, disc_min, , );
rtDeclareVariable(float4, disc_hole, , );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float3, texcoord, attribute texcoord, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );


RT_PROGRAM void intersect(int) {

  /*
   * for disc
   *                 ^ n
   *         i       |
   * --------*-------C--------------------
   *       d/        |----------r-------->|
   *       /
   *      * O
   *
   * Ray direction = d
   * Ray origin = O
   * disc centre = C
   * disc radius = r
   *
   * ray position = ray origin + time * ray direction
   * -> r(t) = O + t * d
   *
   * Ray intersects plane which disc is in if
   * (r(t) - C) . n = 0
   *
   * Ray hits disc is in plane and within radius
   * r(t) - C < r
   * -> (r(t) - C)^2 < r^2 (so handles both directions)
   *
   * t = (ray origin - disc centre) / ray direction  in normalised plane
   * -> t = (O - C).n / d.n
   */

  // Disc properties
  float r = disc_shape.w;
  float3 c = make_float3(disc_shape.x,disc_shape.y,disc_shape.z);
  float3 n = make_float3(0.f,0.f,1.f); // normal

  // Hole properties
  float3 hole_c = make_float3(disc_hole.x, disc_hole.y, disc_hole.z);
  float hole_r = disc_hole.w;

  // ray properties
  float3 o = ray.origin;
  float3 d = ray.direction;

  float rr = radius*radius;
  float hole_rr = hole_radius * hole_radius;

  float mm = dot(m, m) ;
  float nn = dot(n, n) ;
  float nd = dot(n, d) ;   // >0 : ray direction in same hemi as normal
  float md = dot(m, d) ;
  float mn = dot(m, n) ;

  // Remove tmin?
  float t_min = 0.f;

  float t_center = -md/nd ;
  float rsq = t_center*(2.f*mn + t_center*nn) + mm;

  // check hole center now
  float3 hole_m = ray.origin - hole_center;
  float hole_md = dot(hole_m, d);
  float hole_mm = dot(hole_m, hole_m);
  float hole_mn = dot(hole_m, n);
  float hole_t_center = -hole_md/nd;
  float hole_rsq = hole_t_center*(2.f*hole_mn + hole_t_center*nn) + hole_mm;

  // TODO let hole not be in center
  float t_cand = (rsq < rr && hole_rsq > hole_rr) ? t_center : t_min;


  bool valid_isect = t_cand > t_min ;
  if(valid_isect) {
    if( rtPotentialIntersection( t_cand ) ) {
      shading_normal = geometric_normal = normalize(d);
      rtReportIntersection(0);
        }
  }
  return;
}

RT_PROGRAM void bounds(int, float result[6]) {
  optix::Aabb *aabb = (optix::Aabb *)result;
  aabb->set(disc_min, disc_max);
}
