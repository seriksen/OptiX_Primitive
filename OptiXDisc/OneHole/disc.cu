#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_aabb_namespace.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>

using namespace optix;

// Communication Variables
rtDeclareVariable(float4, disc_shape, , );
rtDeclareVariable(float3, disc_max, , );
rtDeclareVariable(float3, disc_min, , );
rtDeclareVariable(float4, disc_hole, , );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float3, texcoord, attribute texcoord, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );


RT_PROGRAM void intersect(int) {

  float radius = disc_shape.w;
  float3 center = make_float3(disc_shape.x,disc_shape.y,disc_shape.z);
  float3 hole_center = make_float3(disc_hole.x, disc_hole.y, disc_hole.z);
  float hole_radius = disc_hole.w;
  float3 m = ray.origin - center;
  float3 n = ray.direction;
  float3 d = make_float3(0.f,0.f,1.f); // normal
  float rr = radius*radius;
  float hole_rr = hole_radius * hole_radius;

  float mm = dot(m, m) ;
  float nn = dot(n, n) ;
  float nd = dot(n, d) ;   // >0 : ray direction in same hemi as normal
  float md = dot(m, d) ;
  float mn = dot(m, n) ;

  float t_min = 0.f;

  float t_center = -md/nd ;
  float rsq = t_center*(2.f*mn + t_center*nn) + mm;

  // check hole center now
  float3 hole_m = ray.origin - hole_center;
  float hole_md = dot(hole_m, d);
  float hole_mm = dot(hole_m, hole_m);
  float hole_mn = dot(hole_m, n);
  float hole_t_center = -hole_md/nd;
  float hole_rsq = hole_t_center*(2.f*hole_mn + hole_t_center*nn) + hole_mm;

  // TODO let hole not be in center
  float t_cand = (rsq < rr && hole_rqs > hole_rr) ? t_center : t_min;


  bool valid_isect = t_cand > t_min ;
  if(valid_isect) {
    if( rtPotentialIntersection( t_cand ) ) {
      shading_normal = geometric_normal = normalize(d);
      rtReportIntersection(0);
        }
  }
  return;
}

RT_PROGRAM void bounds(int, float result[6]) {
  optix::Aabb *aabb = (optix::Aabb *)result;
  aabb->set(disc_min, disc_max);
}
