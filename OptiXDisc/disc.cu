#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_aabb_namespace.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>

using namespace optix;

// Communication Variables
rtDeclareVariable(float3, cylinder_p, , );
rtDeclareVariable(float3, cylinder_q, , );
rtDeclareVariable(float4, cylinder_r, , );
rtDeclareVariable(float3, cylinder_min, , );
rtDeclareVariable(float3, cylinder_max, , );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float3, texcoord, attribute texcoord, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );


RT_PROGRAM void intersect(int) {

  float z1 = 0.01f;
  float z2 = -0.01f;
  float zc = (z1 + z2) / 2.f;
  float radius = 0.1f;
  float inner = 0.01; // ? what is this?
  float dz = (z2 - z1) / 2.f;
  float3 center = make_float3(0.f,0.f,0.f);

  float3 m = ray.origin - center;
  float3 n = ray.direction;
  float3 d = make_float3(0.f,0.f,1.f);

  float rr = radius*radius;
  float ii = inner * inner;

  float mm = dot(m, m) ;
  float nn = dot(n, n) ;
  float nd = dot(n, d) ;   // >0 : ray direction in same hemi as normal
  float md = dot(m, d) ;
  float mn = dot(m, n) ;

  float t_min = 0.f;

  float t_center = -md/nd ;
  float rsq = t_center*(2.f*mn + t_center*nn) + mm  ;   // ( m + tn).(m + tn)

  float t_delta  = nd < 0.f ? -zdelta/nd : zdelta/nd ;    // <-- pragmatic make t_delta +ve

  float root1 = t_center - t_delta ;
  float root2 = t_center + t_delta ;   // root2 > root1

  float t_cand = ( rsq < rr && rsq > ii ) ? ( root1 > t_min ? root1 : root2 ) : t_min ;

  float side = md + t_cand*nd ;

  bool valid_isect = t_cand > t_min ;
  if(valid_isect) {
    if( rtPotentialIntersection( t_cand ) ) {
      shading_normal = geometric_normal = normalize(d);
      rtReportIntersection(0);
        }
  }
  return;
}

RT_PROGRAM void bounds(int, float result[6]) {
  optix::Aabb *aabb = (optix::Aabb *)result;
  aabb->set(cylinder_min, cylinder_max);
}
