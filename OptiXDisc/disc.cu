#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include <optixu/optixu_aabb_namespace.h>

using namespace optix; // Not actually used anywhere here?

// OptiX Communication Variables
rtDeclareVariable(float4, disc_center, , );
rtDeclareVaribale(float3, disc_props, , );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float3, texcoord, attribute texcoord, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );


RT_PROGRAM void intersect(int)
{
  // Declare local variables
  float radius = disc_center.w;
  float3 center = make_float3(disc_center.x, disc_center.y, disc_center.z);
  float z1 = disc_prop.x;
  float z2 = disc_prop.y;
  float zc = (z1 + z2)/2.f;
  float dz = (z1 - z2)/2.f;
  float3 m = ray.origin - center;
  float3 n = ray.direction;
  float3 d = make_float3(0.f, 0.f, 1.0f);
  float mm = dot(m,m);
  float nn = dot(n,n);
  float nd = dot(n,d);
  float md = dot(m,d);
  float mn = dot(m,n);
  float rr = radius * radius;
  float ii = 0.000001f;

  float t_c = -md/nd;
  float rsq = t_c * (2.f * mn + t_c * nn) + mm;
  float dt = nd < 0.f ? -dz/nd : dz/nd;

  float root1 = t_c - dt;
  float root2 = t_c + dt;
  float t_cand = ( rsq < rr && rsq > ii ) ? ( root1 > 0.f ? root1 : root2 ) : 0.f ;

  if (t_cand > 0.f) {
    if(rtPotentialIntersection(t_cand)) {
      shading_normal = geometric_normal = normalize(d);
      rtReportIntersection(0);
    }
  }
}

RT_PROGRAM void bounds (int, float result[6])
{
  float3 disc_p = disc_props;
  float4 disc_c = disc_center;

  float3 bbmin = make_float3(disc_c.x - disc_c.w,
                             disc_c.y - disc_c.w,
                             disc_p.x);
  float3 bbmax = make_float3(disc_c.x + disc_c.w,
                             disc_c.y + disc_c.w,
                             disc_p.y);


  optix::Aabb* aabb = (optix::Aabb*)result;
  aabb->set(bbmin, bbmax);
}
