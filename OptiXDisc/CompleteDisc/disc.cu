#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_aabb_namespace.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>

using namespace optix;

// Communication Variables
rtDeclareVariable(float4, disc_shape, , );
rtDeclareVariable(float3, disc_max, , );
rtDeclareVariable(float3, disc_min, , );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float3, texcoord, attribute texcoord, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );


RT_PROGRAM void intersect(int) {


  /*
   * for disc
   *                 ^ n
   *         i       |
   * --------*-------C--------------------
   *       d/        |----------r-------->|
   *       /
   *      * O
   *
   * Ray direction = d
   * Ray origin = O
   * disc centre = C
   * disc radius = r
   *
   * ray position = ray origin + time * ray direction
   * -> r(t) = O + t * d
   *
   * Ray intersects plane which disc is in if
   * (r(t) - C) . n = 0
   *
   * Ray hits disc is in plane and within radius
   * r(t) - C < r
   * -> (r(t) - C)^2 < r^2 (so handles both directions)
   *
   * t = (ray origin - disc centre) / ray direction  in normalised plane
   * -> t = - (O - C).n / d.n
   */

  // Disc properties
  float r = disc_shape.w;
  float3 c = make_float3(disc_shape.x,disc_shape.y,disc_shape.z);
  float3 n = make_float3(0.f,0.f,1.f); // normal

  // ray properties
  float3 d = ray.direction;
  float3 o = ray.origin;

  // t
  float t = - dot((o - c), n) / dot(d, n);

  // check if intersects
  float rt_sqrt = t * (2.f * dot((o-c), d) + t * dot(d,d)) + dot(o-c,o-c);
  float rr = r*r;

  if (rt_sqrt < rr && t > 0.f) {
    if (rtPotentialIntersection(t)) {
      shading_normal = geometric_normal = normalize(n);
      rtReportIntersection(0);
    }
  }
  return;
}

RT_PROGRAM void bounds(int, float result[6]) {
  optix::Aabb *aabb = (optix::Aabb *)result;
  aabb->set(disc_min, disc_max);
}
